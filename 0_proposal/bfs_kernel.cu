#include "hip/hip_runtime.h"
__global__ void
BFS_kernel( ... ) 
{
  if(threadIdx.x < NUM_BIN){
    q.reset(threadIdx.x, blockDim);
  }
  __syncthreads();

  tid = blockIdx.x*THREADS_PER_BLOCK
		...  + threadIdx.x;
  if( tid < no_of_nodes)
  {
    visit_node(q1[tid], threadIdx.x 
	& MOD_OP, q, overflow,
       	g_color, g_cost, gray_shade);
  }
  __syncthreads();

  if(threadIdx.x == 0){
    ...
    shift = atomicAdd(tail,tot_sum);
  }
  __syncthreads();
}

