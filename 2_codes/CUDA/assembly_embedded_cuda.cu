/*	Cuda program with inline ptx assembly to test how information can be
	passed from hihg level cuda code to PTX assembly (which can thereafter
	be instrumented by GPGPU-SIM				

	Author	: Archit Gupta
	Date	: November 18, 2015
	Purpose	: CS252 course project
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add_array(float* in_a, float* in_b, float* out_c, int N)
{
	int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	__asm__("EXTRN:");
	if(idx < N)
	{
		__asm__("INTRN:");
		if(idx < N/2)
		{
			out_c[idx] = in_a[idx] + in_b[idx];
			__asm__("INTRN:");
		}
		else
		{
			out_c[idx] = in_b[idx] - in_a[idx];
		}
	}
}

int main(void)
{
	const int NUM_ELEMENTS = 16;
	const int BLOCK_SIZE = 16;
	const int NUM_BLOCKS = (NUM_ELEMENTS/BLOCK_SIZE) + ((NUM_ELEMENTS%BLOCK_SIZE == 0) ? 0:1);
	size_t FLOAT_ARRAY_SIZE = sizeof(float)*NUM_ELEMENTS;

	float *h_a, *h_b, *h_c;			// data arrays on the host machine
	float *d_a, *d_b, *d_c;			// data arrays on the GPU
	
	h_a = (float *) malloc(FLOAT_ARRAY_SIZE); // Allocating memory for the data	
	h_b = (float *) malloc(FLOAT_ARRAY_SIZE); // on the host machine	
	h_c = (float *) malloc(FLOAT_ARRAY_SIZE); 
 
	hipMalloc((void**) &d_a, FLOAT_ARRAY_SIZE);
	hipMalloc((void**) &d_b, FLOAT_ARRAY_SIZE);
	hipMalloc((void**) &d_c, FLOAT_ARRAY_SIZE);
	
	// Initializing data in the input arrays. Lets initialize
	// a[i] = 2i;
	// b[i] = i^2+1; 
	
	for(int i=0; i<NUM_ELEMENTS; i++)
	{
		h_a[i] = (float) 2*i;
		h_b[i] = (float) i*i+1;	
	}

	// We expect out function to produce 
	// c[i] = (i+1)^2 if i<N/2
	// c[i] = (i-1)^2 otherwise
	
	hipMemcpy(d_a, h_a, FLOAT_ARRAY_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, FLOAT_ARRAY_SIZE, hipMemcpyHostToDevice);

	add_array <<< NUM_BLOCKS, BLOCK_SIZE >>> (d_a, d_b, d_c, NUM_ELEMENTS);

	hipMemcpy(h_c, d_c, FLOAT_ARRAY_SIZE, hipMemcpyDeviceToHost);

	for(int i=0; i<NUM_ELEMENTS; i++)
	{
		printf("%d:a[i]=%f, b[i]=%f, c[i]=%f\n", i, h_a[i], h_b[i], h_c[i]); 
	}
	
	return 0;
}
