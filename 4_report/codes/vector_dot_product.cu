
#include <hip/hip_runtime.h>
__device__ float reduction_sum; 

__global__ 
void vector_dot(float* a, float* b,
			 float* c, int N)
{
   int idx = (blockIdx.x + blockDim.x) 
			+ threadIdx.x;
   if (idx < N)
   {
      c[idx] = a[idx] * b[idx];	
   }
   __syncthreads();
   if (idx == 0)
   {	
      for (unsigned i=0; i < N; i++)
      { 
         reduction_sum += c[idx];
      }
   }		
}
