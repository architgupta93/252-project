#include "hip/hip_runtime.h"
__device__ float reduction_sum; 

__global__  vector_dot(float* a, float* b, float* c, int N)
{
	float  dot = 0;
	int idx = (blockIdx.x + blockDim.x) + threadIdx.x;
	__asm__("EXTRN:");
	if (idx < N)
	{
		c[idx] = a[idx] * b[idx];	
	}
	__syncthreads();
	__asm__("EXTRN:");
	if (idx == 0)
	{	
		__asm__("INTRN:");
		for (unsigned i=0; i < N; i++)
		{
			reduction_sum += c[idx];
		}
	}		
}
